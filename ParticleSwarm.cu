// ParticleSwarm.cpp : This file contains the 'main' function. Program execution begins and ends there.
//

#include "OP.h"
#include "Rastriging.h"
#include "Particle.h"
#include "Swarm.h"
#include "XM.h"

/*
To set CUDA working with Visual Studio 2017:
- right click on project
- build dependencies -> build customization
- tick on some installed CUDA version
- OK
and it should build.
*/

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>
#include <string>
#include <stdio.h> 

static void testSwarm(OP &op) {
	bool useCuda = true;
	bool CUDAposvel = false;
	XM xm;
	// TODO: Create particles based on warp size?
	const int size = 20;
	const int dim = 2; // OP decision space dimension
	Swarm swarm(size, dim, op);
	int generations = 1000;

	std::cout << "First generation:" << std::endl;
	swarm.print();

	for(int i = 0; i < size; i++) {
		for(size_t j = 1; j < dim; ++j) {
			std::cout << "(" << swarm.xx[i*j - 1 + j] << "," << swarm.xx[i*j + j] << ") -> ("
				<< swarm.vv[i*j - 1 + j] << "," << swarm.vv[i*j + j] << ")" << std::endl;
		}
	}

	xm.startSwarm(std::chrono::high_resolution_clock::now());

	for(int i = 0; i < generations; ++i) {
		swarm.updateParticlePositions(CUDAposvel);
	}
	xm.endSwarm(std::chrono::high_resolution_clock::now());
	swarm.end();


	for(int i = 0; i < size; i++) {
		for(size_t j = 1; j < dim; ++j) {
			std::cout << "(" << swarm.xx[i*j - 1 + j] << "," << swarm.xx[i*j + j] << ") -> (" 
				<< swarm.vv[i*j - 1 + j] << "," << swarm.vv[i*j + j] << ")" << std::endl;
		}
	}

	std::cout << "Last generation:" << std::endl;
	swarm.print();


	std::cout << "-- " << op.name << " problem --" << std::endl;
	std::cout << "Running the swarm took: " << xm.swarmDuration << " s." << std::endl;
	std::cout << "of which " << std::endl;
	std::cout << "initialization      " << swarm.initTimeMicS << " micro seconds." << std::endl;
	std::cout << "updating best value " << swarm.updateBestTimeMicS << " micro seconds." << std::endl;
	std::cout << "updating particles  " << swarm.updateParticlesTimeMicS << " micro seconds." << std::endl;
	std::cout << "of which" << std::endl;
	std::cout << "updating positions       " << swarm.updatePosTimeMicS << " micro seconds." << std::endl;
	std::cout << "updating velocities      " << swarm.updateVelTimeMicS << " micro seconds." << std::endl;
	std::cout << "updating function values " << swarm.updateFunTimeMicS << " micro seconds." << std::endl;
	std::cout << "Total function evaluations " << swarm.fEvals << std::endl;

	// Print out csv-style to be pasted in Excel
	char sep = ';';
	// Few words how this version is different from base version.
	std::string upgrade = "Base";
	std::cout 
		<< std::fixed 
		<< upgrade << sep 
		<< generations << sep
		<< size << sep
		<< xm.swarmDuration * 1000000 << sep
		<< swarm.initTimeMicS << sep
		<< swarm.updateBestTimeMicS << sep
		<< swarm.updateParticlesTimeMicS << sep
		<< swarm.updatePosTimeMicS << sep
		<< swarm.updateVelTimeMicS << sep
		<< swarm.updateFunTimeMicS << sep
		<< swarm.fEvals 
		<< std::endl;
}

int main() {
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for(int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
	}

	srand(145623);
	Rastriging problem;
	testSwarm(problem);
}
