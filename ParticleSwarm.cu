// ParticleSwarm.cpp : This file contains the 'main' function. Program execution begins and ends there.
//

#include "OP.h"
#include "Rastriging.h"
#include "Particle.h"
#include "Swarm.h"
#include "XM.h"

/*
To set CUDA working with Visual Studio 2017:
- right click on project
- build dependencies -> build customization
- tick on some installed CUDA version
- OK
and it should build.
*/

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>
#include <string>
#include <stdio.h> 


static void testRastriging(OP &op) {
	std::vector<std::vector<float>> range = op.getSearchRange();
	std::cout << "Search range:" << std::endl;

	for(int i = 0; i < range.size(); i++) {
		std::cout << range[i][0] << " <= " << "x" << i + 1 << " <= " << range[i][1] << std::endl;
	}
	std::vector<float> point = { 0.0f,0.0f };
	std::cout << "f(" << point[0] << "," << point[1] << ") = " << op.evaluate(point) << std::endl;
}

static void testParticle(OP &op) {
	std::vector<float> x = { 0.f,0.f }; // val should be 0 at (0,0)
	std::vector<float> v = { 0.2f,0.1f };
	Particle part(x, v, op);
	// Print in initial position
	part.print();
	// Move the particle around for a few times
	int maxRounds = 2000;
	// Print every now and then
	int printEvery = 100;
	// Guide the particle towards origin at all times
	std::vector<float> dir = { 0.f,0.f };

	/*for(int i = 0; i < maxRounds; i++) {
		part.update(dir);
		if(i % printEvery == 0) {
			part.print();
		}
	}*/

	part.print();
}


static void testSwarm(OP &op) {
	bool useCuda = true;
	XM xm;
	// TODO: Create particles based on warp size?
	const int size = 20;
	const int dim = 2; // OP decision space dimension
	Swarm swarm(size, dim, op);
	int generations = 1000;

	std::cout << "First generation:" << std::endl;
	swarm.print();


	xm.startSwarm(std::chrono::high_resolution_clock::now());
	//if(useCuda){
	for(int i = 0; i < generations; ++i) {
		hipError_t cudaStatus = hipGetLastError();
		if(cudaStatus != hipSuccess) {
			fprintf(stderr, "PS before update failed: %s\n", hipGetErrorString(cudaStatus));
		}
		//swarm.updateParticlePositions<<<1,1>>>();
		swarm.updateParticlePositions();
		cudaStatus = hipGetLastError();
		if(cudaStatus != hipSuccess) {
			fprintf(stderr, "PS after update failed: %s\n", hipGetErrorString(cudaStatus));
		}
		// Wait for GPU to finish before accessing on host
		hipDeviceSynchronize();
	}
	/*} else {
		swarm.updateParticlePositions();
	}*/
	xm.endSwarm(std::chrono::high_resolution_clock::now());
	swarm.end();


	for(int i = 0; i < size; i++) {
		for(size_t j = 1; j < dim; ++j) {
			std::cout << "(" << swarm.xx[i*j - 1 + j] << "," << swarm.xx[i*j + j] << ")" << std::endl;
		}
	}

	std::cout << "Last generation:" << std::endl;
	swarm.print();


	std::cout << "-- " << op.name << " problem --" << std::endl;
	std::cout << "Running the swarm took: " << xm.swarmDuration << " s." << std::endl;
	std::cout << "of which " << std::endl;
	std::cout << "initialization      " << swarm.initTimeMicS << " micro seconds." << std::endl;
	std::cout << "updating best value " << swarm.updateBestTimeMicS << " micro seconds." << std::endl;
	std::cout << "updating particles  " << swarm.updateParticlesTimeMicS << " micro seconds." << std::endl;
	std::cout << "of which" << std::endl;
	std::cout << "updating positions       " << swarm.updatePosTimeMicS << " micro seconds." << std::endl;
	std::cout << "updating velocities      " << swarm.updateVelTimeMicS << " micro seconds." << std::endl;
	std::cout << "updating function values " << swarm.updateFunTimeMicS << " micro seconds." << std::endl;
	std::cout << "Total function evaluations " << swarm.fEvals << std::endl;

	// Print out csv-style to be pasted in Excel
	char sep = ';';
	// Few words how this version is different from base version.
	std::string upgrade = "Base";
	std::cout 
		<< std::fixed 
		<< upgrade << sep 
		<< generations << sep
		<< size << sep
		<< xm.swarmDuration * 1000000 << sep
		<< swarm.initTimeMicS << sep
		<< swarm.updateBestTimeMicS << sep
		<< swarm.updateParticlesTimeMicS << sep
		<< swarm.updatePosTimeMicS << sep
		<< swarm.updateVelTimeMicS << sep
		<< swarm.updateFunTimeMicS << sep
		<< swarm.fEvals 
		<< std::endl;
}

int main() {
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for(int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
	}

	srand(145623);
	Rastriging problem;
	testSwarm(problem);
}
